#include <cmath>
#include <string>
#include <cstdio>
#include <vector>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <hipblas.h>
#include <hip/hip_runtime.h>



// функция активации
__global__ void nn_Sigmoid(float* arr, int size)
{
	int id = threadIdx.x;

	arr[id] = 1 / (1 + exp(-arr[id]));
}



class NN
{
private:
	hipblasHandle_t handle;
	float alpha, beta;
	float* weights, * biases, * output;
	int input_size, output_size;
	bool activation_true;

	// считывание весов из файла
	void read_weights(std::string pathToWeights) {
		float* host_array = new float[input_size * output_size];
		float* host_array_row = new float[input_size * output_size];

		std::ifstream fin(pathToWeights);
		for (int i = 0; i < input_size * output_size; i++) fin >> host_array_row[i];
		fin.close();

		for (int i = 0; i < input_size; i++) {
			for (int j = 0; j < output_size; j++) {
				host_array[i * output_size + j] = host_array_row[((j) * (input_size)) + (i)];
			}
		}
		hipMalloc(&weights, output_size * input_size * sizeof(float));
		hipMemcpy(weights, host_array, output_size * input_size * sizeof(float), hipMemcpyHostToDevice);
		delete[] host_array, host_array_row;
	};

	// считывание добавочных членов из файла
	void read_biases(std::string pathToWeights) {
		float* host_array = new float[output_size];

		std::ifstream fin(pathToWeights);
		for (int i = 0; i < output_size; i++) fin >> host_array[i];
		fin.close();

		hipMalloc(&biases, output_size * sizeof(float));
		hipMemcpy(biases, host_array, output_size * sizeof(float), hipMemcpyHostToDevice);
		delete[] host_array;
	};

public:
	// конструкторы
	NN() {
		input_size = 0;
		output_size = 0;
		alpha = 1.0;
		beta = 1.0;
		activation_true = true;
	};

	NN(std::string pathToWeights, std::string pathToBiases, int inSize, int outSize, bool activation) {
		alpha = 1.0;
		beta = 1.0;
		input_size = inSize;
		output_size = outSize;
		read_weights(pathToWeights);
		read_biases(pathToBiases);
		activation_true = activation;

	};


	// линейный слой
	float* Linear(float* input) {
		hipblasCreate(&handle);
		hipblasSgemv(handle, HIPBLAS_OP_N, output_size, input_size, &alpha, weights, output_size, input, 1, &beta, biases, 1);
		hipblasDestroy(handle);
		if (activation_true) {
			nn_Sigmoid <<<1, output_size>>> (biases, output_size);
		}
		return biases;
	};

	// деструктор
	~NN() {
		if (weights != nullptr) hipFree(weights);
		if (biases != nullptr) hipFree(biases);
	};
};

///////////  MODEL   /////////////
class Net
{
private:
	float* array;
	int input_size, output_size;
	std::vector<NN> layers;

	// чтение input
	void read_input(std::string pathToWeights) {
		float* host_array = new float[input_size];

		std::ifstream fin(pathToWeights);
		for (int i = 0; i < input_size; i++) fin >> host_array[i];
		fin.close();


		hipMalloc(&array, input_size * sizeof(float));
		hipMemcpy(array, host_array, input_size * sizeof(float), hipMemcpyHostToDevice);
		delete[] host_array;
	};

	// вывод
	void print_result(float* arr) {
		float* host_array = new float[output_size];
		hipMemcpy(host_array, arr, output_size * sizeof(float), hipMemcpyDeviceToHost);
		
		std::cout << "Result: " << std::endl;

		std::cout << host_array[0] << std::endl;

		//проврка
		float pattern = round(0.5757734179496765 * 100) / 100;
		host_array[0] = round(host_array[0] * 100) / 100;

		if (pattern == host_array[0]) std::cout << "IT'S RIGHT ANSWER!" << std::endl;
		else std::cout << "result(" << host_array[0] << ") and pattern(" << pattern << ")" << std::endl;

		delete[] host_array;
	};

public:
	// конструктор по умолчанию
	Net() {
		input_size = 1024;
		output_size = 1;
	};

	// запуск базовой сети
	void forward(std::string pathToFile) {
		read_input(pathToFile);
		NN layer1("weights1.bin", "biases1.bin", 1024, 256, true);
		array = layer1.Linear(array);
		NN layer2("weights2.bin", "biases2.bin", 256, 16, true);
		array = layer2.Linear(array);
		NN layer3("weights3.bin", "biases3.bin", 16, 1, true);
		array = layer3.Linear(array);

		print_result(array);
	}

	// деструктор
	~Net() {
		if (array != nullptr) hipFree(array);//нулевой указатель 
	};
};

int main()
{
	Net model;
	model.forward("input.bin");
	return 0;
}
